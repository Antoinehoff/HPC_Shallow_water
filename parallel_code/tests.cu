#include "hip/hip_runtime.h"
#include "tests.cuh"
#include "kernels.cuh"
#include "functions.h"
#include <iostream>
#include <iomanip>
#include <fstream>
#include <cmath>
#include <string>
#include <sstream>


void test_max(){
  cout << "\t -----\tTesting maximum CUDA function\t-----\t" << '\n';
  int numElements   = 2001*2001;
  size_t memsize    = numElements * sizeof(double);
  double CPU_answer = -1;

  double *h_A       = (double *)malloc(memsize);
  double *GPU_answer  = (double *)malloc(sizeof(double));

  load_initial_state("../data/Data_nx2001_500km_T0.2_h.bin",   h_A,    numElements);

  clock_t timer_h;
  timer_h=clock();
  for(int i=0; i<numElements; i++){
    CPU_answer = fmaxf(CPU_answer, h_A[i]);
  }
  timer_h = clock()-timer_h;
  timer_h = (double) timer_h/CLOCKS_PER_SEC*1000.0;

  double *d_A, *d_answer; int *d_mutex;
  hipMalloc((void **)&d_A,memsize);
  hipMalloc((void **)&d_answer,sizeof(double));
  hipMemset(d_answer, 0, sizeof(float));
  hipMalloc((void **)&d_mutex, sizeof(int));
	hipMemset(d_mutex, 0, sizeof(float));

  int threadsPerBlock = 128;
  int blocksPerGrid   = 256;//(numElements + threadsPerBlock - 1)/threadsPerBlock;
  cout << "blocksPerGrid x threadsPerBlock :" << threadsPerBlock << " x "
       << blocksPerGrid << " (= " << threadsPerBlock * blocksPerGrid  << ")\n";
  float gpu_elapsed_time;
  hipEvent_t gpu_start, gpu_stop;
  hipEventCreate(&gpu_start);
  hipEventCreate(&gpu_stop);

  hipMemcpy(d_A, h_A, memsize, hipMemcpyHostToDevice);

  hipEventRecord(gpu_start, 0);
  find_maximum_kernel<<<threadsPerBlock,blocksPerGrid>>>(d_A, d_answer, d_mutex, numElements);
  hipEventRecord(gpu_stop, 0);

  hipMemcpy(GPU_answer, d_answer, sizeof(double), hipMemcpyDeviceToHost);

	hipEventSynchronize(gpu_stop);
	hipEventElapsedTime(&gpu_elapsed_time, gpu_start, gpu_stop);
	hipEventDestroy(gpu_start);
	hipEventDestroy(gpu_stop);

  std::cout  << "The gpu took: "        <<  gpu_elapsed_time      <<  " milli-seconds"  <<  std::endl;
  std::cout  << "The cpu took: "        <<  timer_h               <<  " milli-seconds"  <<  std::endl;
  if(abs(GPU_answer[0]-CPU_answer) < 1e-5){
    std::cout <<  "Test PASSED "        << '\n';
  }
  else{
    std::cout <<  "Test FAILED "        << '\n';
    std::cout  << " CPU Answer : "      <<  CPU_answer            << '\n';
    std::cout  << " GPU Answer : "      <<  GPU_answer[0]           << '\n';
  }


  hipFree(d_A); hipFree(d_answer); hipFree(d_mutex);
  free(h_A); free(GPU_answer);

}

void test_update_dt(){
  cout << "\t -----\tTesting update_dt CUDA function\t-----\t" << '\n';
  int nx            = 2001;
  int Size          = 500;
  double dx         = Size/nx;
  int numElements   = 2001*2001;
  size_t memsize    = numElements * sizeof(double);
  double CPU_dt     = -1;

  double *H         = (double *)malloc(memsize);
  double *HU        = (double *)malloc(memsize);
  double *HV        = (double *)malloc(memsize);
  double *GPU_dt      = (double *)malloc(sizeof(double));

  load_initial_state("../data/Data_nx2001_500km_T0.2_h.bin",  H,  numElements);
  load_initial_state("../data/Data_nx2001_500km_T0.2_hu.bin", HU, numElements);
  load_initial_state("../data/Data_nx2001_500km_T0.2_hv.bin", HV, numElements);

  clock_t timer_h;
  timer_h   =clock();
  CPU_dt    = update_dt(H,HU,HV,dx,numElements);
  timer_h   = clock()-timer_h;
  timer_h   = (double) timer_h/CLOCKS_PER_SEC*1000.0;

  double *d_H, *d_HU, *d_HV, *d_dt;
  hipMalloc((void **)&d_H,memsize);
  hipMalloc((void **)&d_HU,memsize);
  hipMalloc((void **)&d_HV,memsize);
  hipMalloc((void **)&d_dt,sizeof(double));

  int threadsPerBlock = 256;
  int blocksPerGrid   = ceil(numElements*1.0/256);//(numElements + threadsPerBlock - 1)/threadsPerBlock;
  cout << "blocksPerGrid x threadsPerBlock :" << threadsPerBlock << " x "
       << blocksPerGrid << " (= " << threadsPerBlock * blocksPerGrid  << ")\n";
  float gpu_elapsed_time;
  hipEvent_t gpu_start, gpu_stop;
  hipEventCreate(&gpu_start);
  hipEventCreate(&gpu_stop);

  hipMemcpy(d_H,  H,  memsize, hipMemcpyHostToDevice);
  hipMemcpy(d_HU, HU, memsize, hipMemcpyHostToDevice);
  hipMemcpy(d_HV, HV, memsize, hipMemcpyHostToDevice);

  hipEventRecord(gpu_start, 0);
  //update_dt_kernel<<<threadsPerBlock,blocksPerGrid>>>(d_H, d_HU, d_HV, d_dt, dx, numElements);
  hipEventRecord(gpu_stop, 0);

  hipMemcpy(GPU_dt, d_dt, sizeof(double), hipMemcpyDeviceToHost);

	hipEventSynchronize(gpu_stop);
	hipEventElapsedTime(&gpu_elapsed_time, gpu_start, gpu_stop);
	hipEventDestroy(gpu_start);
	hipEventDestroy(gpu_stop);

  std::cout  << "The gpu took: "        <<  gpu_elapsed_time      <<  " milli-seconds"  <<  std::endl;
  std::cout  << "The cpu took: "        <<  timer_h               <<  " milli-seconds"  <<  std::endl;
  if(abs(GPU_dt[0] - CPU_dt) < 1e-5){
    std::cout <<  "Test PASSED "        << '\n';
  }
  else{
    std::cout <<  "Test FAILED "        << '\n';
    std::cout  << " CPU Answer : "      <<  CPU_dt              << '\n';
    std::cout  << " GPU Answer : "      <<  GPU_dt[0]           << '\n';
  }


  hipFree(d_H); hipFree(d_HU); hipFree(d_HV); hipFree(d_dt);
  free(H); free(HU); free(HV); free(GPU_dt);
}
