#include "hip/hip_runtime.h"
#include "tests.cuh"
#include "kernels.cuh"
#include "functions.h"
#include <iostream>
#include <iomanip>
#include <fstream>
#include <cmath>
#include <string>
#include <sstream>


void test_max(){
  cout << "\t -----\tTesting maximum CUDA function\t-----\t" << '\n';
  int numElements   = 2001*2001;
  size_t memsize    = numElements * sizeof(double);
  double CPU_answer = -1;

  double *h_A       = (double *)malloc(memsize);
  double *h_answer  = (double *)malloc(sizeof(double));

  load_initial_state("../data/Data_nx2001_500km_T0.2_h.bin",   h_A,    numElements);

  clock_t timer_h;
  timer_h=clock();
  for(int i=0; i<numElements; i++){
    CPU_answer = fmaxf(CPU_answer, h_A[i]);
  }
  timer_h = clock()-timer_h;
  timer_h = (double) timer_h/CLOCKS_PER_SEC*1000.0;

  double *d_A, *d_answer; int *d_mutex;
  hipMalloc((void **)&d_A,memsize);
  hipMalloc((void **)&d_answer,sizeof(double));
  hipMemset(d_answer, 0, sizeof(float));
  hipMalloc((void **)&d_mutex, sizeof(int));
	hipMemset(d_mutex, 0, sizeof(float));

  int threadsPerBlock = 256;
  int blocksPerGrid   = ceil(numElements*1.0/256);//(numElements + threadsPerBlock - 1)/threadsPerBlock;
  cout << "blocksPerGrid x threadsPerBlock :" << threadsPerBlock << " x "
       << blocksPerGrid << " (= " << threadsPerBlock * blocksPerGrid  << ")\n";
  float gpu_elapsed_time;
  hipEvent_t gpu_start, gpu_stop;
  hipEventCreate(&gpu_start);
  hipEventCreate(&gpu_stop);

  hipMemcpy(d_A, h_A, memsize, hipMemcpyHostToDevice);

  hipEventRecord(gpu_start, 0);
  find_maximum_kernel<<<threadsPerBlock,blocksPerGrid>>>(d_A, d_answer, d_mutex, numElements);
  hipEventRecord(gpu_stop, 0);

  hipMemcpy(h_answer, d_answer, sizeof(double), hipMemcpyDeviceToHost);

	hipEventSynchronize(gpu_stop);
	hipEventElapsedTime(&gpu_elapsed_time, gpu_start, gpu_stop);
	hipEventDestroy(gpu_start);
	hipEventDestroy(gpu_stop);

  std::cout  << "The gpu took: "        <<  gpu_elapsed_time      <<  " milli-seconds"  <<  std::endl;
  std::cout  << "The cpu took: "        <<  timer_h               <<  " milli-seconds"  <<  std::endl;
  if(h_answer[0]-CPU_answer < 1e-5){
    std::cout <<  "Test PASSED "        << '\n';
  }
  else{
    std::cout <<  "Test FAILED "        << '\n';
    std::cout  << " CPU Answer : "      <<  CPU_answer            << '\n';
    std::cout  << " GPU Answer : "      <<  h_answer[0]           << '\n';
  }


  hipFree(d_A); hipFree(d_answer); hipFree(d_mutex);
  free(h_A); free(h_answer);

}
