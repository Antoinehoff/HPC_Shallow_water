#include "hip/hip_runtime.h"
#include<iostream>
#include "kernels.cuh"

using namespace std;


int main(){
  int numElements   = 1<<22;
  size_t memsize    = numElements * sizeof(double);
  double CPU_answer = -1;

  double *h_A       = (double *)malloc(memsize);
  double *h_answer  = (double *)malloc(sizeof(double));

  for (int i = 0; i < numElements; ++i){
    h_A[i] = numElements*double(rand())/RAND_MAX;
  }

  clock_t timer_h;
  timer_h=clock();
  CPU_answer = find_maximum_CPU(h_A, numElements);
  timer_h = clock()-timer_h;
  timer_h = (double) timer_h/CLOCKS_PER_SEC*1000.0;

  double *d_A, *d_answer; int *d_mutex;
  hipMalloc((void **)&d_A,memsize);
  hipMalloc((void **)&d_answer,sizeof(double));
  hipMemset(d_answer, 0, sizeof(float));
  hipMalloc((void **)&d_mutex, sizeof(int));
	hipMemset(d_mutex, 0, sizeof(float));

  dim3 threadsPerBlock = 256;
  dim3 blocksPerGrid   = 256; //(numElements + threadsPerBlock - 1)/threadsPerBlock;

  float gpu_elapsed_time;
  hipEvent_t gpu_start, gpu_stop;
  hipEventCreate(&gpu_start);
  hipEventCreate(&gpu_stop);

  hipMemcpy(d_A, h_A, memsize, hipMemcpyHostToDevice);

  hipEventRecord(gpu_start, 0);
  find_maximum_kernel<<<threadsPerBlock,blocksPerGrid>>>(d_A, d_answer, d_mutex, numElements);
  hipEventRecord(gpu_stop, 0);

  hipMemcpy(h_answer, d_answer, sizeof(double), hipMemcpyDeviceToHost);

	hipEventSynchronize(gpu_stop);
	hipEventElapsedTime(&gpu_elapsed_time, gpu_start, gpu_stop);
	hipEventDestroy(gpu_start);
	hipEventDestroy(gpu_stop);

  std::cout  << "The gpu took: "  <<  gpu_elapsed_time  <<  " milli-seconds"  <<  std::endl;
  std::cout  << " GPU Answer : "  <<  h_answer[0]       << '\n';
  std::cout  << "The cpu took: "  <<  timer_h           <<  " milli-seconds"  <<  std::endl;
  std::cout  << " CPU Answer : "  <<  CPU_answer        << '\n';



  hipFree(d_A); hipFree(d_answer); hipFree(d_mutex);
  free(h_A); free(h_answer);

  return 0;
}
