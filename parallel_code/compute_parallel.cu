#include "hip/hip_runtime.h"
/*
Parallel version in CUDA/C++ of compute.cpp code
*/
#include <iostream>
#include <iomanip>
#include <fstream>
#include <cmath>
#include <cstring>
#include <sstream>
#include "kernels.cuh" //parallel kernels
#include "functions.h" //Sequential functions

using namespace std;

int main(){
  // Basic Parameters of the simulation :
  clock_t timer = clock();
  int     Size        = 500;                          // Size of map, Size*Size [km]
  size_t  nx          = 2001;                         // Grid 1D size
  float   Tend        = 0.2;                          // Simulation time in hours [hr]
  double  dx          = ((float)Size)/((float)nx);    // Grid spacening
  int     numElements = nx*nx;                        // Total number of elements
  size_t  memsize     = numElements * sizeof(double); // Memory size of one array
  int     Ntmax       = 250;                            // Choose the maximum of iteration
  // Simulation variables HOST
  double  T           = 0.0;                          // Time
  int     nt          = 0;                            // Iteration counter
  double  dt          = 0.;                           // Time step
  double  C           = 0.0;                          // Coefficient 1/2*dt/dx
  double *H,    *HU,  *HV;                            // Water height and x,y speeds
  double *Ht,   *HUt, *HVt;                           // Temporary memory of H HU and HV
  double *Zdx,  *Zdy;                                 // Topology of the map
  // Simulation variables DEVICE
  double *d_H,    *d_HU,  *d_HV;                      // Water height and x,y speeds
  double *d_Ht,   *d_HUt, *d_HVt;                     // Temporary memory of H HU and HV
  double *d_Zdx,  *d_Zdy;                             // Topology of the map
  // Tracking variables
  double *dt_array;                                   // Record the evolution time steps
  string  datapath     = "../data/";                  // Path for the data

  // Allocate host memory for loading the initial conditions
  cout << " Allocating host memory .."  <<  endl;
  H   = (double *)malloc(memsize);
  HU  = (double *)malloc(memsize);  HV  = (double *)malloc(memsize);
  Ht  = (double *)malloc(memsize);
  HUt = (double *)malloc(memsize);  HVt = (double *)malloc(memsize);
  Zdx = (double *)malloc(memsize);  Zdy = (double *)malloc(memsize);
  dt_array = (double *)malloc(Ntmax*sizeof(double));

  // Load initial state on host memory
  load_initial_data(H, HU, HV, Zdx, Zdy, datapath, nx, Size, Tend, numElements);

  // Allocate device memory for computing
  cout  <<  " Allocating device memory on host.." <<  endl;
  hipMalloc((void **)  &d_H,   memsize);
  hipMalloc((void **)  &d_HU,  memsize); hipMalloc((void **)  &d_HV, memsize);
  hipMalloc((void **)  &d_Ht,  memsize);
  hipMalloc((void **)  &d_HUt, memsize); hipMalloc((void **)  &d_HVt,memsize);
  hipMalloc((void **)  &d_Zdx, memsize); hipMalloc((void **)  &d_Zdy,memsize);

  // Copy initial conditions from host to device
  cout << " Copying variables from host to device.."  <<  endl;
  hipMemcpy(d_H,   H,    memsize,  hipMemcpyHostToDevice);
  hipMemcpy(d_HU,  HU,   memsize,  hipMemcpyHostToDevice);
  hipMemcpy(d_HV,  HV,   memsize,  hipMemcpyHostToDevice);
  hipMemcpy(d_Zdx, Zdx,  memsize,  hipMemcpyHostToDevice);
  hipMemcpy(d_Zdy, Zdy,  memsize,  hipMemcpyHostToDevice);
  hipMemcpy(d_Ht,  Ht,   memsize,  hipMemcpyHostToDevice);
  hipMemcpy(d_HUt, HUt,  memsize,  hipMemcpyHostToDevice);
  hipMemcpy(d_HVt, HVt,  memsize,  hipMemcpyHostToDevice);

  // One dimensional grid block threads version :
  int Nthreadx = 128;
  dim3 threadsPerBlock(Nthreadx);
  int Nblockx = ceil(nx*nx*1.0/Nthreadx);
  dim3 numBlocks(Nblockx);
  cout << "1D parallel model description :"<<endl;
  cout <<"\t Number of elements \t\t:" << nx*nx << endl;
  cout <<"\t Number of blocks needed \t:" << Nblockx << "x" << 1 << endl;
  cout <<"\t Nthreads \t\t\t:" << Nblockx*Nthreadx <<endl;

  // Evolution loop
  while (T < Tend and nt < Ntmax) {
        // Compute the time-step length
        dt = update_dt(H,HU,HV,dx,numElements);
        if(T+dt > Tend){
          dt = Tend-T;
        }
        //Print status
        cout  << " Computing for T=" << T+dt << " ("<< 100*(T+dt)/Tend << "%), "
              <<  "dt="             << dt   << endl;
        // Copy solution to temp storage and enforce boundary condition
        //cpy_to(Ht,H,numElements);
        //cpy_to(HUt,HU,numElements);
        //cpy_to(HVt,HV,numElements);
        //enforce_BC(Ht, HUt, HVt, nx);
        // Compute a time-step
        C = (.5*dt/dx);
        //FV_time_step(H,HU,HV,Zdx,Zdy,Ht,HUt,HVt,C,dt,nx);
        FV_time_step_kernel<<<Nblockx,Nthreadx>>>(d_H,d_HU,d_HV,d_Zdx,d_Zdy,d_Ht,d_HUt,d_HVt,C,dt,nx);
        // Impose tolerances
        //impose_tolerances(Ht,HUt,HVt,numElements);
        if(nt < Ntmax) dt_array[nt]=dt;
        T = T + dt;
        nt++;
  }

  // Copy device result to the host memory
  cout << " Copy the output data from the CUDA device to the host memory" << endl;
  hipMemcpy(Ht, d_Ht, memsize, hipMemcpyDeviceToHost);
  // Save solution to disk
  ostringstream soutfilename;
  soutfilename <<"../output/CUDA_Solution_nx"<<to_string(nx)<<"_"<<to_string(Size)<<"km_T"<<Tend<<"_h.bin"<< setprecision(2);
  string outfilename = soutfilename.str();

  ofstream fout;
  fout.open(outfilename, std::ios::out | std::ios::binary);
  cout<<" Writing solution in "<<outfilename<<endl;
  fout.write(reinterpret_cast<char*>(&Ht[0]), numElements*sizeof(double));
  fout.close();

  //save dt historic
  ostringstream soutfilename2;
  soutfilename2 <<"../output/CUDA_dt_nx"<<to_string(nx)<<"_"<<to_string(Size)<<"km_T"<<Tend<<"_h.bin"<< setprecision(2);
  outfilename = soutfilename2.str();
  fout.open(outfilename, std::ios::out | std::ios::binary);
  cout<<" Writing solution in "<<outfilename<<endl;
  fout.write(reinterpret_cast<char*>(&dt_array[0]), Ntmax*sizeof(double));
  fout.close();

  // Free device global memory
  cout  <<  " Free device memory space.." <<  endl;
  hipFree(d_H);    hipFree(d_HU);   hipFree(d_HV);   hipFree(d_Zdx);
  hipFree(d_Zdy);  hipFree(d_Ht);   hipFree(d_HUt);  hipFree(d_HVt);

  // Free host memory
  cout  <<  " Free host memory space.." <<  endl;
  free(H);    free(HU);   free(HV);   free(Zdx); free(Zdy);
  free(Ht);   free(HUt);  free(HVt);  free(dt_array);

  // Timer end
  timer = clock()-timer;
  timer = (double)(timer)/CLOCKS_PER_SEC*1000;
  cout  <<  "Ellapsed time : "  <<  timer/60000  <<  "min "
        <<  timer/1000  <<  "s " << timer%1000 << "ms" << endl;
  return 0;
}
