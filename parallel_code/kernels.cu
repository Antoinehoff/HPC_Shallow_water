#include "hip/hip_runtime.h"
#include "kernels.cuh"

__global__ void FV_time_step_kernel(double *d_H, double *d_HU, double *d_HV,
const double *d_Zdx, const double *d_Zdy, double *d_Ht, double *d_HUt,
double *d_HVt, double C, double dt, int nx){

	unsigned int idx = threadIdx.x + blockIdx.x*blockDim.x;
	unsigned int x = idx/nx;
	unsigned int y = idx%nx;

	//Check if the thread is inside the domain
	if(x < nx and y < nx){
		//COPY LAST STATE IN TEMPORARY VARIABLES
		if(x < nx and y < nx){
			d_Ht[idx] 	= d_H[idx];
			d_HUt[idx]	= d_HUt[idx];
			d_HVt[idx]	= d_HVt[idx];
		}
		__syncthreads();

		//ENFORCE BC
		if(x == 0 or x == nx-1 or y == 0 or y == nx-1){
			d_Ht [y * (nx) + x]  = d_Ht [y * (nx) + x];
			d_HUt[y * (nx) + x]  = d_HUt[y * (nx) + x];
			d_HVt[y * (nx) + x]  = d_HVt[y * (nx) + x];
		}
		__syncthreads();

		//FINITE VOLUME STEP
		if(x > 0 and y > 0 and x < nx-1 and y < nx-1){
			d_H[y * (nx) + x]=
				0.25*( d_Ht[y * (nx) + (x+1)]+d_Ht[y * (nx) + (x-1)]
							+d_Ht[(y+1) * (nx) + x]+d_Ht[(y-1) * (nx) + x])+
				C   *( d_HUt[(y-1) * (nx) + x]-d_HUt[(y+1) * (nx) + x]
							+d_HVt[y * (nx) + (x-1)]-d_HVt[y * (nx) + (x+1)]);

			d_HU[y * (nx) + x]=
				0.25*( d_HUt[y * (nx) + (x+1)]+d_HUt[y * (nx) + (x-1)]
							+d_HUt[(y+1) * (nx) + x]+d_HUt[(y-1) * (nx) + x])
							-dt*g*d_H[y * (nx) + x]*d_Zdx[y * (nx) + x]
			 +C   *( pow(d_HUt[(y-1) * (nx) + x],2)/d_Ht[(y-1) * (nx) + x]
							+0.5*g*pow(d_Ht[(y-1) * (nx) + x],2)
							-pow(d_HUt[(y+1) * (nx) + x],2)/d_Ht[(y+1) * (nx) + x]
							-0.5*g*pow(d_Ht[(y+1) * (nx) + x],2))
			 +C   *( d_HUt[y * (nx) + (x-1)]*d_HVt[y * (nx) + (x-1)]/d_Ht[y * (nx) + (x-1)]
							-d_HUt[y * (nx) + (x+1)]*d_HVt[y * (nx) + (x+1)]/d_Ht[y * (nx) + (x+1)]);

			d_HV[y * (nx) + x]  =
				0.25*( d_HVt[y * (nx) + (x+1)]+d_HVt[y * (nx) + (x-1)]
							+d_HVt[(y+1) * (nx) + x]+d_HVt[(y-1) * (nx) + x])
							-dt*g*d_H[y * (nx) + x]*d_Zdy[y * (nx) + x]
			 +C   *( d_HUt[(y-1) * (nx) + x]*d_HVt[(y-1) * (nx) + x]/d_Ht[(y-1) * (nx) + x]
							-d_HUt[(y+1) * (nx) + x]*d_HVt[(y+1) * (nx) + x]/d_Ht[(y+1) * (nx) + x])
			 +C   *( pow(d_HVt[y * (nx) + (x-1)],2)/d_Ht[y * (nx) + (x-1)]
							+0.5*g*pow(d_Ht[y * (nx) + (x-1)],2)
							-pow(d_HVt[y * (nx) + (x+1)],2)/d_Ht[y * (nx) + (x+1)]
							-0.5*g*pow(d_Ht[y * (nx) + (x+1)],2));
		}
		__syncthreads();

		//IMPOSING TOLERANCES
		if(d_Ht[idx]<0){
			d_Ht[idx] = 1e-5;
		}
		if(d_Ht[idx] <= 1e-5){
			d_HUt[idx] = 0;
			d_HVt[idx] = 0;
		}
	}
}

__global__ void find_maximum_device(double *array, double *max, int *mutex, unsigned int numElements)
{
	unsigned int index = threadIdx.x + blockIdx.x*blockDim.x;
	if(index<numElements){
		unsigned int stride = gridDim.x*blockDim.x;
		unsigned int offset = 0;
		__shared__ double cache[256];
		double temp = -1.0;
		while(index + offset < numElements){
			temp = fmaxf(temp, array[index + offset]);
			offset += stride;
		}
		cache[threadIdx.x] = temp;
		__syncthreads();
		// reduction
		unsigned int i = blockDim.x/2;
		while(i != 0){
			if(threadIdx.x < i){
				cache[threadIdx.x] = fmaxf(cache[threadIdx.x], cache[threadIdx.x + i]);
			}
			__syncthreads();
			i /= 2;
		}
		if(threadIdx.x == 0){
			while(atomicCAS(mutex,0,1) != 0);  //lock
			*max = fmaxf(*max, cache[0]);
			atomicExch(mutex, 0);  //unlock
		}
	}
}

__global__ void find_maximum_kernel(double *array, double *max, int *mutex, unsigned int numElements)
{
	unsigned int index = threadIdx.x + blockIdx.x*blockDim.x;
	unsigned int stride = gridDim.x*blockDim.x;
	unsigned int offset = 0;
	__shared__ double cache[256];
	double temp = -1.0;
	while(index + offset < numElements){
		temp = fmaxf(temp, array[index + offset]);
		offset += stride;
	}
	cache[threadIdx.x] = temp;
	__syncthreads();
	// reduction
	unsigned int i = blockDim.x/2;
	while(i != 0){
		if(threadIdx.x < i){
			cache[threadIdx.x] = fmaxf(cache[threadIdx.x], cache[threadIdx.x + i]);
		}
		__syncthreads();
		i /= 2;
	}
	if(threadIdx.x == 0){
		while(atomicCAS(mutex,0,1) != 0);  //lock
		*max = fmaxf(*max, cache[0]);
		atomicExch(mutex, 0);  //unlock
	}
}


__device__ void update_dt_kernel(const double *H, const double *HU, const double *HV,
                                 double* dt,      double dx,        int numElements){
   //Compute the max of mu and give dt back
   double mu = 0.0;
   double newmu = 0.0;
   for(int i=0; i<numElements; i++){
     newmu = sqrt(pow(max(abs(HU[i]/H[i]-sqrt(H[i]*g)),abs(HU[i]/H[i]+sqrt(H[i]*g))),2)
                 +pow(max(abs(HV[i]/H[i]-sqrt(H[i]*g)),abs(HV[i]/H[i]+sqrt(H[i]*g))),2));
     if(newmu > mu){
       mu = newmu;
       }
     }
     *dt = dx/(sqrt(2.0)*mu);
}
